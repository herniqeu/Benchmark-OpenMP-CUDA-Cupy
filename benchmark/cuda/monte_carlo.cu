
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void monte_carlo_kernel(int n_points, int* inside, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_points) {
        hiprand_init(1234, idx, 0, &states[idx]);
        float x = hiprand_uniform(&states[idx]);
        float y = hiprand_uniform(&states[idx]);
        if (x*x + y*y <= 1.0f) atomicAdd(inside, 1);
    }
}

extern "C" float monte_carlo_pi(int n_points) {
    int* d_inside;
    int h_inside = 0;
    hiprandState* d_states;
    
    hipMalloc(&d_inside, sizeof(int));
    hipMalloc(&d_states, n_points * sizeof(hiprandState));
    hipMemcpy(d_inside, &h_inside, sizeof(int), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int num_blocks = (n_points + block_size - 1) / block_size;
    monte_carlo_kernel<<<num_blocks, block_size>>>(n_points, d_inside, d_states);
    
    hipMemcpy(&h_inside, d_inside, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_inside);
    hipFree(d_states);
    
    return 4.0f * h_inside / n_points;
}